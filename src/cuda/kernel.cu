#include "hip/hip_runtime.h"
#include "kernel.cuh"

uint64_t lastInteractionUpdate; 		/**< Date of the last interraction update */
struct vector3_t * cu_position;			// Float array of entities positions [x0,y0,z0][x1,y1,z1]...
struct vector3_t * cu_currVelocity;		// Float array of current entities velocities [vx0,vy0,vz0][vx1,vy1,vz1]...
struct vector3_t * cu_newVelocity;		// Float array of new entities velocities [vx0,vy0,vz0][vx1,vy1,vz1]...
float * cu_mass;						// Float array 

/**
 * Each thread of each block has recieve an entity 
 * and compute its new velocity by comparing it to all others entities
 * @param cu_position: vector of all the positions of the entity of the universe
 * @param cu_newVelocity: new velocity computed by each thread
 * @param cu_mass: vector of all masses of the entities of the universe
 * @param entityCount: number of entity in the universe (and size of the above vectors)
 */
__global__ void kernel_compute_new_velocity(struct vector3_t * cu_position, struct vector3_t * cu_newVelocity, float * cu_mass, uint32_t entityCount)
{
	uint32_t curEntity = fmaf(blockIdx.x, blockDim.x, threadIdx.x);	
	uint32_t entityIndex;
	struct vector3_t delta;		/** Vector from the current entity to the neighbor */
	float distance;				/** Distance between the current entity to the neighbor */
	float coeff;				/** Coefficient applied for velocity calculation */
	float curX, curY, curZ;	

	if (curEntity >= entityCount) {
		return;
	}

	/** Reset the velocity */
	(cu_newVelocity[curEntity]).x = 0;
	(cu_newVelocity[curEntity]).y = 0;
	(cu_newVelocity[curEntity]).z = 0;

	curX = (cu_position[curEntity]).x;
	curY = (cu_position[curEntity]).y;
	curZ = (cu_position[curEntity]).z;

	for (entityIndex = 0; entityIndex < entityCount; entityIndex++) {

		/** Ignore the current entity */
		if (entityIndex == curEntity) {
			continue;
		}

		/** Compute the slope vector delta */
		delta.x = (cu_position[entityIndex]).x - curX;
		delta.y = (cu_position[entityIndex]).y - curY;
		delta.z = (cu_position[entityIndex]).z - curZ;

		/** Compute the distance by computing the abs of delta */
		distance = sqrtf(POW_2(delta.x) + POW_2(delta.y) + POW_2(delta.z));

		if (distance == 0) {
			// TODO Entity collisions
			continue;		
		}

		/** Pre-Computed coeff to avoid repetition */
		coeff = cu_mass[entityIndex] / (POW_3(distance));

		/** Add the neighboor contribution to the new velocity of the current entity */
		(cu_newVelocity[curEntity]).x = fmaf(delta.x, coeff, (cu_newVelocity[curEntity]).x);
		(cu_newVelocity[curEntity]).y = fmaf(delta.y, coeff, (cu_newVelocity[curEntity]).y);
		(cu_newVelocity[curEntity]).z = fmaf(delta.z, coeff, (cu_newVelocity[curEntity]).z);
	}
}

__global__ void kernel_apply_new_velocity(struct vector3_t * cu_position, 
		struct vector3_t * cu_currVelocity, struct vector3_t * cu_newVelocity, 
		float constFactor, float si_timeFactor, int entityCount) 
{
	uint32_t curEntity = fmaf(blockIdx.x, blockDim.x, threadIdx.x);
	float curVelX, curVelY, curVelZ;

	// Avoid memory overrun
	if (curEntity >= entityCount) {
		return;
	}

	// Save values in local memory
	curVelX = (cu_currVelocity[curEntity]).x;
	curVelY = (cu_currVelocity[curEntity]).y;
	curVelZ = (cu_currVelocity[curEntity]).z;

	// Update current velocity
	cu_currVelocity[curEntity].x = fmaf(cu_newVelocity[curEntity].x, constFactor, curVelX);
	cu_currVelocity[curEntity].y = fmaf(cu_newVelocity[curEntity].y, constFactor, curVelY);
	cu_currVelocity[curEntity].z = fmaf(cu_newVelocity[curEntity].z, constFactor, curVelZ);

	// Update position
	cu_position[curEntity].x = fmaf(si_timeFactor, curVelX, cu_position[curEntity].x);
	cu_position[curEntity].y = fmaf(si_timeFactor, curVelY, cu_position[curEntity].y);
	cu_position[curEntity].z = fmaf(si_timeFactor, curVelZ, cu_position[curEntity].z);

}

/**
 *	Allocate memory on GPU and copy init value
 */
uint32_t cuda_allocate(struct universe_t *universe) 
{	
	cuda_lib_malloc((void**) &cu_position, universe->entityCount * sizeof(struct vector3_t));
	cuda_lib_malloc((void**) &cu_currVelocity, universe->entityCount * sizeof(struct vector3_t));
	cuda_lib_malloc((void**) &cu_newVelocity, universe->entityCount * sizeof(struct vector3_t));
	cuda_lib_malloc((void**) &cu_mass, universe->entityCount * sizeof(float));

	cuda_lib_memcpy(cu_position, universe->position, universe->entityCount * sizeof(struct vector3_t), hipMemcpyHostToDevice);
	cuda_lib_memcpy(cu_currVelocity ,universe->currVelocity, universe->entityCount * sizeof(struct vector3_t), hipMemcpyHostToDevice);
	cuda_lib_memcpy(cu_mass, universe->mass, universe->entityCount * sizeof(float), hipMemcpyHostToDevice);

	return 0;
}

/**
 * Init the lastInteractionUpdate variable to avoid jump on start 
 */
void cuda_start_make_a_move(void)
{
	lastInteractionUpdate = getTimeOfDayInUs();
}

/**
 * Compute the next move of all entities
 */
void cuda_make_a_move(struct universe_t *universe, float constFactor, float si_timeFactor) 
{
	int nbBlock = 1;
	int nbThreadPerBlock = 1023;
	float deltaInSec;

	/** Compute new velocities on all threads of all blocks */
	kernel_compute_new_velocity<<<nbBlock, nbThreadPerBlock>>>(cu_position, cu_newVelocity, cu_mass, universe->entityCount);

	/** Compute the time between two apply and update the constant factor */
	deltaInSec = (getTimeOfDayInUs() - lastInteractionUpdate) / 1000000.0f;
	lastInteractionUpdate = getTimeOfDayInUs();
	constFactor *= deltaInSec;
	si_timeFactor *= deltaInSec;

	/** Apply new velocity */
	kernel_apply_new_velocity<<<nbBlock, nbThreadPerBlock>>>(cu_position, cu_currVelocity, cu_newVelocity, constFactor, si_timeFactor, universe->entityCount);

	/** Copy results from device to host */ 
	cuda_lib_memcpy(universe->position, cu_position, universe->entityCount * sizeof(struct vector3_t), hipMemcpyDeviceToHost);
}

/**
 * Free all memory previously allocated
 */
void cuda_free(void) 
{
	cuda_lib_free((void *)cu_position);
	cuda_lib_free((void *)cu_currVelocity);
	cuda_lib_free((void *)cu_newVelocity);
	cuda_lib_free((void *)cu_mass);
}
